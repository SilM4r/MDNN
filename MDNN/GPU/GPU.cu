#include <hip/hip_runtime.h>


extern "C" __declspec(dllexport) void LayerCalculation(float* a, float* b, float* c, float* result, int size, int quantity);

// nvcc -shared -o gpu.dll GPU.cu
// Kernel funkce pro sčítání vektorů
__global__ void LayerCalculationKernel(float* values, float* weights,float* bias , float* result, int size, int quantity) {
    float sum = 0.0f;  
    int i = threadIdx.x + blockDim.x * blockIdx.x; 

    if(i < quantity) 
    {  
        for (int k = 0; k < size; k++) {
            sum += values[k] * weights[i * size + k];   
        }

        result[i] = sum + bias[i]; 
    }
}


// Exportovaná funkce, která bude volána z C#
extern "C" __declspec(dllexport) void LayerCalculation(float* values, float* weights, float* bias, float* result, int size, int quantity) {
    float* d_values;
    float* d_weights;
    float* d_bias;
    float* d_result;

    // Alokace paměti na GPU
    hipMalloc((void**)&d_values, size * sizeof(float));
    hipMalloc((void**)&d_bias, quantity * sizeof(float));
    hipMalloc((void**)&d_weights, size * quantity * sizeof(float));
    hipMalloc((void**)&d_result, quantity * sizeof(float));

    // Kopírování dat z CPU do GPU
    hipMemcpy(d_values, values, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, quantity * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, size * quantity * sizeof(float), hipMemcpyHostToDevice);

    // Volání kernelu s 256 vlákny na blok
    int blockSize = 256;
    int numBlocks = (quantity + blockSize - 1) / blockSize;
    LayerCalculationKernel<<<numBlocks, blockSize>>>(d_values, d_weights,d_bias, d_result, size, quantity);

    // Kopírování výsledků zpět na CPU
    hipMemcpy(result, d_result, quantity * sizeof(float), hipMemcpyDeviceToHost);

    // Uvolnění paměti na GPU
    hipFree(d_values);
    hipFree(d_weights);
    hipFree(d_bias);
    hipFree(d_result);
}